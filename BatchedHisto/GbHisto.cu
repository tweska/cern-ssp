#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include <iomanip>

#include "CUDAHelpers.cuh"
#include "types.h"

#include "GbHisto.h"

using std::chrono::high_resolution_clock;
using std::chrono::duration_cast;
using std::chrono::duration;
using std::chrono::milliseconds;

/// @brief Increase a bin in the histogram by a certain weight.
template <typename T>
__device__ inline void AddBinContent(T *histogram, usize bin, f64 weight) {
    atomicAdd(&histogram[bin], (T)weight);
}

/// @brief Find the corresponding bin in a histogram axis based on a given x value.
__device__ inline usize FindBin(f64 x, const f64 *binEdges, isize binEdgesIdx, usize nBins, f64 xMin, f64 xMax) {
    if (x < xMin)
        return 0;
    if (!(x < xMax))
        return nBins + 1;

    if (binEdgesIdx < 0)
        return 1 + usize(nBins * (x - xMin) / (xMax - xMin));
    return 1 + ROOT::Experimental::CUDAHelpers::BinarySearch(nBins + 1, &binEdges[binEdgesIdx], x);
}

/// @brief Find the corresponding bin in a histogram based on a coordinate.
__device__ inline usize GetBin(
    usize i, usize nDims,
    f64 *binEdges, isize *binEdgesIdx, usize *nBinsAxis,
    f64 *xMin, f64 *xMax,
    f64 *coords, usize bulkSize
) {
    usize bin = 0;
    for (isize d = nDims - 1; d >= 0; --d) {
        f64 *x = &coords[d * bulkSize];
        usize binD = FindBin(x[i], binEdges, binEdgesIdx[d], nBinsAxis[d] - 2, xMin[d], xMax[d]);
        bin = bin * nBinsAxis[d] + binD;
    }
    return bin;
}

/// @brief Global memory batch histogram kernel.
template <typename T>
__global__ void HistogramGlobal(
    T *histograms, usize *histoResultOffset, usize *histoOffset, usize nHistos,
    f64 *binEdges, isize *binEdgesOffset, usize *nBinsAxis, usize *nDims,
    f64 *xMin, f64 *xMax,
    f64 *coords, f64 *weights, usize bulkSize
) {
    usize tid = threadIdx.x + blockDim.x * blockIdx.x;
    usize stride = blockDim.x * gridDim.x;

    for (usize i = tid; i < bulkSize * nHistos; i += stride) {
        usize h = i / bulkSize;
        usize hoff = histoOffset[h];  // Histogram Offset

        T *histogram = &histograms[histoResultOffset[h]];

        usize bin = GetBin(
            i % bulkSize, nDims[h],
            binEdges, &binEdgesOffset[hoff], &nBinsAxis[hoff],
            &xMin[hoff], &xMax[hoff],
            &coords[hoff * bulkSize], bulkSize
        );

        if (weights)
            AddBinContent<T>(histogram, bin, weights[i]);
        else
            AddBinContent<T>(histogram, bin, 1.0);
    }
}

template<typename T, usize BlockSize>
GbHisto<T, BlockSize>::GbHisto(
    usize nHistos, const usize *nDims, const usize *nBinsAxis,
    const f64 *xMin, const f64 *xMax,
    const f64 *binEdges, const isize *binEdgesOffset,
    usize maxBulkSize,
    Timer<> *rtInit, Timer<> *rtTransfer, Timer<> *rtKernel, Timer<> *rtResult
) {
    if (rtInit) rtInit->start();

    this->nHistos = nHistos;
    this->maxBulkSize = maxBulkSize;

    nBins = 0;
    nAxis = 0;
    h_histoResultOffset = new usize[nHistos];
    h_histoOffset = new usize[nHistos];

    for (usize h = 0, i = 0; h < nHistos; ++h) {
        h_histoResultOffset[h] = nBins;
        nAxis += nDims[h];
        h_histoOffset[h] = i;
        usize nInterBins = 1;
        for (usize d = 0; d < nDims[h]; ++d, ++i) {
            nInterBins *= nBinsAxis[i];
        }
        nBins += nInterBins;
    }

    ERRCHECK(hipMalloc(&d_histograms, sizeof(T) * nBins));
    if (binEdges) {
        ERRCHECK(hipMalloc(&d_binEdges, sizeof(f64) * nBins));
    } else {
        d_binEdges = nullptr;
    }
    ERRCHECK(hipMalloc(&d_xMin, sizeof(f64) * nAxis));
    ERRCHECK(hipMalloc(&d_xMax, sizeof(f64) * nAxis));
    ERRCHECK(hipMalloc(&d_nDims, sizeof(usize) * nHistos));
    ERRCHECK(hipMalloc(&d_nBinsAxis, sizeof(usize) * nAxis));
    ERRCHECK(hipMalloc(&d_histoResultOffset, sizeof(usize) * nHistos));
    ERRCHECK(hipMalloc(&d_histoOffset, sizeof(usize) * nHistos));
    ERRCHECK(hipMalloc(&d_binEdgesOffset, sizeof(isize) * nAxis));
    ERRCHECK(hipMalloc(&d_coords, sizeof(f64) * nAxis * maxBulkSize));
    ERRCHECK(hipMalloc(&d_weights, sizeof(f64) * nHistos * maxBulkSize));
    hipDeviceSynchronize();

    ERRCHECK(hipMemset(d_histograms, 0, sizeof(T) * nBins));
    if (d_binEdges) {
        ERRCHECK(hipMemcpy(d_binEdges, binEdges, sizeof(f64) * nBins, hipMemcpyHostToDevice));
    }
    ERRCHECK(hipMemcpy(d_xMin, xMin, sizeof(f64) * nAxis, hipMemcpyHostToDevice));
    ERRCHECK(hipMemcpy(d_xMax, xMax, sizeof(f64) * nAxis, hipMemcpyHostToDevice));
    ERRCHECK(hipMemcpy(d_nDims, nDims, sizeof(usize) * nHistos, hipMemcpyHostToDevice));
    ERRCHECK(hipMemcpy(d_nBinsAxis, nBinsAxis, sizeof(usize) * nAxis, hipMemcpyHostToDevice));
    ERRCHECK(hipMemcpy(d_histoResultOffset, h_histoResultOffset, sizeof(usize) * nHistos, hipMemcpyHostToDevice));
    ERRCHECK(hipMemcpy(d_histoOffset, h_histoOffset, sizeof(usize) * nHistos, hipMemcpyHostToDevice));
    ERRCHECK(hipMemcpy(d_binEdgesOffset, binEdgesOffset, sizeof(isize) * nAxis, hipMemcpyHostToDevice));
    hipDeviceSynchronize();

    if (rtInit) rtInit->pause();
    this->rtInit = rtInit;
    this->rtTransfer = rtTransfer;
    this->rtKernel = rtKernel;
    this->rtResult = rtResult;
}

template<typename T, usize BlockSize>
GbHisto<T, BlockSize>::~GbHisto() {
    ERRCHECK(hipFree(d_histograms));
    ERRCHECK(hipFree(d_binEdges));
    ERRCHECK(hipFree(d_xMin));
    ERRCHECK(hipFree(d_xMax));
    ERRCHECK(hipFree(d_nDims));
    ERRCHECK(hipFree(d_nBinsAxis));
    ERRCHECK(hipFree(d_histoResultOffset));
    ERRCHECK(hipFree(d_histoOffset));
    ERRCHECK(hipFree(d_binEdgesOffset));
    ERRCHECK(hipFree(d_coords));
    ERRCHECK(hipFree(d_weights));

    delete[] h_histoResultOffset;
    delete[] h_histoOffset;
}

template <typename T, usize BlockSize>
void GbHisto<T, BlockSize>::RetrieveResults(T *histograms, f64 *stats) {
    if (rtResult) rtResult->start();

    ERRCHECK(hipMemcpy(histograms, d_histograms, sizeof(T) * nBins, hipMemcpyDeviceToHost));
    ERRCHECK(hipDeviceSynchronize());

    if (rtResult) rtResult->pause();
}

template<typename T, usize BlockSize>
void GbHisto<T, BlockSize>::FillN(usize n, const f64 *coords, const f64 *weights) {
    if (n > maxBulkSize) {
        FillN(maxBulkSize, coords, weights);
        if (weights)
            FillN(n - maxBulkSize, coords + maxBulkSize, weights + maxBulkSize);
        else
            FillN(n - maxBulkSize, coords + maxBulkSize, nullptr);
        return;
    }

    if (rtTransfer) rtTransfer->start();

    ERRCHECK(hipMemcpy(d_coords, coords, sizeof(f64) * nAxis * n, hipMemcpyHostToDevice));

    f64 *weightsPtr = nullptr;
    if (weights) {
        ERRCHECK(hipMemcpy(d_weights, weights, sizeof(f64) * n * nHistos, hipMemcpyHostToDevice));
        weightsPtr = d_weights;
    }

    if (rtTransfer) rtTransfer->pause();
    if (rtKernel) rtKernel->start();

    usize nThreads = nHistos * n;
    usize nBlocks = nThreads / BlockSize + (nThreads % BlockSize != 0);

    HistogramGlobal<T><<<nBlocks, BlockSize>>>(
        d_histograms, d_histoResultOffset, d_histoOffset, nHistos,
        d_binEdges, d_binEdgesOffset, d_nBinsAxis, d_nDims,
        d_xMin, d_xMax,
        d_coords, weightsPtr, n
    );
    ERRCHECK(hipPeekAtLastError());

    if (rtKernel) rtKernel->pause();
}


template class GbHisto<f64>;
