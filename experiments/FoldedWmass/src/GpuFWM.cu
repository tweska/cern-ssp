#include "hip/hip_runtime.h"
#include "CUDAHelpers.cuh"

#include "GpuFWM.h"

#define ISOLATION_CRITICAL 0.5

__device__
inline void AddBinContent(f64 *histogram, const usize bin)
{
    atomicAdd(&histogram[bin], 1.0f);
}

__device__
inline usize FindBin(const f64 x, const usize nBins, const f64 xMin, const f64 xMax)
{
    if (x < xMin)
        return 0;
    if (!(x < xMax))
        return nBins - 1;

    return 1 + static_cast<usize>((nBins - 2) * (x - xMin) / (xMax - xMin));
}

__device__
inline f32 angle(
    const f32 x1, const f32 y1, const f32 z1,
    const f32 x2, const f32 y2, const f32 z2
) {
    // cross product
    const f32 cx = y1 * z2 - y2 * z1;
    const f32 cy = x1 * z2 - x2 * z1;
    const f32 cz = x1 * y2 - x2 * y1;

    // norm of cross product
    const f32 c = sqrt(cx * cx + cy * cy + cz * cz);

    // dot product
    const f32 d = x1 * x2 + y1 * y2 + z1 * z2;

    return atan2(c, d);
}

__device__
inline f32 invariantMassPxPyPzM(
   const f32 x1, const f32 y1, const f32 z1, const f32 mass1,
   const f32 x2, const f32 y2, const f32 z2, const f32 mass2
) {
    // Numerically stable computation of Invariant Masses
    const f32 pp1 = x1 * x1 + y1 * y1 + z1 * z1;
    const f32 pp2 = x2 * x2 + y2 * y2 + z2 * z2;

    if (pp1 <= 0 && pp2 <= 0)
        return (mass1 + mass2);
    if (pp1 <= 0) {
        f32 mm = mass1 + sqrt(mass2*mass2 + pp2);
        f32 m2 = mm*mm - pp2;
        return m2 >= 0 ? sqrt(m2) : sqrt(-m2);
    }
    if (pp2 <= 0) {
        f32 mm = mass2 + sqrt(mass1*mass1 + pp1);
        f32 m2 = mm*mm - pp1;
        return m2 >= 0 ? sqrt(m2) : sqrt(-m2);
    }

    const f32 mm1 =  mass1 * mass1;
    const f32 mm2 =  mass2 * mass2;

    const f32 r1 = mm1 / pp1;
    const f32 r2 = mm2 / pp2;
    const f32 x = r1 + r2 + r1 * r2;
    const f32 a = angle(x1, y1, z1, x2, y2, z2);
    const f32 cos_a = cos(a);
    f32 y;
    if (cos_a >= 0){
        y = (x + sin(a) * sin(a)) / (sqrt(x + 1) + cos_a);
    } else {
        y = sqrt(x + 1) - cos_a;
    }

    const f32 z = 2.0f * sqrt(pp1 * pp2);

    // Return invariant mass with (+, -, -, -) metric
    return sqrt(mm1 + mm2 + y * z);
}

__device__
inline f32 invariantMassPxPyPzE(
   const f32 x1, const f32 y1, const f32 z1, const f32 e1,
   const f32 x2, const f32 y2, const f32 z2, const f32 e2
) {
    const f32 pp1 = x1 * x1 + y1 * y1 + z1 * z1;
    const f32 pp2 = x2 * x2 + y2 * y2 + z2 * z2;

    const f32 mm1 = e1 * e1 - pp1;
    const f32 mm2 = e2 * e2 - pp2;

    const f32 mass1 = (mm1 >= 0) ? sqrt(mm1) : 0;
    const f32 mass2 = (mm2 >= 0) ? sqrt(mm2) : 0;

    return invariantMassPxPyPzM(x1, y1, z1, mass1, x2, y2, z2, mass2);
}

__device__
inline f32 invariantMassPtEtaPhiE(
    const f32 pt1, const f32 eta1, const f32 phi1, const f32 e1,
    const f32 pt2, const f32 eta2, const f32 phi2, const f32 e2
) {
    const f32 x1 = pt1 * cos(phi1);
    const f32 y1 = pt1 * sin(phi1);
    const f32 z1 = pt1 * sinh(eta1);

    const f32 x2 = pt2 * cos(phi2);
    const f32 y2 = pt2 * sin(phi2);
    const f32 z2 = pt2 * sinh(eta2);

    return invariantMassPxPyPzE(x1, y1, z1, e1, x2, y2, z2, e2);
}

__device__
inline f32 forwardFolding(
    const f32 recoPt,
    const f32 truePt,
    const f32 s,
    const f32 r
) {
    return s * recoPt + (recoPt - truePt) * (r - s);
}

__device__
f32 foldedMass(
    f32 recoPt1, const f32 recoEta1, const f32 recoPhi1, const f32 recoE1,
    f32 recoPt2, const f32 recoEta2, const f32 recoPhi2, const f32 recoE2,
    const f32 truePt1, const f32 truePt2,
    const f32 scale, const f32 resolution
) {
    // Apply forward folding if both truePt values are valid.
    if (truePt1 >= 0 && truePt2 >= 0) {
        recoPt1 = forwardFolding(recoPt1, truePt1, scale, resolution);
        recoPt2 = forwardFolding(recoPt2, truePt2, scale, resolution);
    }

    // Return Invariant mass of sum.
    return invariantMassPtEtaPhiE(
        recoPt1, recoEta1, recoPhi1, recoE1,
        recoPt2, recoEta2, recoPhi2, recoE2
    ) / 1e3f;
}

__global__
void FillKernel(
    f64 *histos, const usize nBins,
    const f64 xMin, const f64 xMax,
    const f32 *scales, const usize nScales,
    const f32 *resolutions, const usize nResolutions,
    const DefCoords *defCoords, const usize bulkSize
) {
    const usize tid = threadIdx.x + blockDim.x * blockIdx.x;
    const usize stride = blockDim.x * gridDim.x;

    for (usize k = tid; k < bulkSize; k += stride) {
        const DefCoords cur = defCoords[k];
        for (usize i = 0; i < nScales; ++i) {
            for (usize j = 0; j < nResolutions; ++j) {
                const f64 mass = foldedMass(
                    cur.recoPt1, cur.recoEta1, cur.recoPhi1, cur.recoE1,
                    cur.recoPt2, cur.recoEta2, cur.recoPhi2, cur.recoE2,
                    cur.truePt1, cur.truePt2,
                    scales[i], resolutions[j]
                );
                const usize bin = FindBin(mass, nBins, xMin, xMax);
                AddBinContent(&histos[(i * nScales + j) * nBins], bin);
            }
        }
    }
}

template <usize BlockSize, usize MaxBulkSize>
GpuFWM<BlockSize, MaxBulkSize>::GpuFWM(
    usize nBins,
    f64 xMin, f64 xMax,
    f32 *scales, usize nScales,
    f32 *resolutions, usize nResolutions,
    Timer<> *rtTransfer, Timer<> *rtKernel, Timer<> *rtResult
) {
    this->nBins = nBins + 2;
    this->xMin = xMin;
    this->xMax = xMax;
    this->nScales = nScales;
    this->nResolutions = nResolutions;

    ERRCHECK(hipMalloc(&d_histos, sizeof(f64) * nScales * nResolutions * this->nBins));
    ERRCHECK(hipMalloc(&d_scales, sizeof(f32) * nScales));
    ERRCHECK(hipMalloc(&d_resolutions, sizeof(f32) * nResolutions));
    ERRCHECK(hipMalloc(&d_defCoords, sizeof(DefCoords) * MaxBulkSize));

    ERRCHECK(hipMemcpy(d_scales, scales, sizeof(f32) * nScales, hipMemcpyHostToDevice));
    ERRCHECK(hipMemcpy(d_resolutions, resolutions, sizeof(f32) * nResolutions, hipMemcpyHostToDevice));
    ERRCHECK(hipDeviceSynchronize());

    this->rtTransfer = rtTransfer;
    this->rtKernel = rtKernel;
    this->rtResult = rtResult;
}

template <usize BlockSize, usize MaxBulkSize>
GpuFWM<BlockSize, MaxBulkSize>::~GpuFWM()
{
    ERRCHECK(hipFree(d_histos));
    ERRCHECK(hipFree(d_scales));
    ERRCHECK(hipFree(d_resolutions));
    ERRCHECK(hipFree(d_defCoords));
}

template <usize BlockSize, usize MaxBulkSize>
void GpuFWM<BlockSize, MaxBulkSize>::RetrieveResult(const usize i, f64 *histograms)
{
    if (rtResult) rtResult->start();
    ERRCHECK(hipMemcpy(histograms, d_histos + i * nBins, sizeof(f64) * nBins, hipMemcpyDeviceToHost));
    ERRCHECK(hipDeviceSynchronize());
    if (rtResult) rtResult->pause();
}

template <usize BlockSize, usize MaxBulkSize>
void GpuFWM<BlockSize, MaxBulkSize>::RetrieveResults(f64 *histograms)
{
    if (rtResult) rtResult->start();
    ERRCHECK(hipMemcpy(histograms, d_histos, sizeof(f64) * nScales * nResolutions * nBins, hipMemcpyDeviceToHost));
    ERRCHECK(hipDeviceSynchronize());
    if (rtResult) rtResult->pause();
}

template <usize BlockSize, usize MaxBulkSize>
void GpuFWM<BlockSize, MaxBulkSize>::FillN(const usize n, const DefCoords *defCoords)
{
    assert(n <= MaxBulkSize);

    if (rtTransfer) rtTransfer->start();
    ERRCHECK(hipMemcpy(d_defCoords, defCoords, sizeof(DefCoords) * n, hipMemcpyHostToDevice));
    ERRCHECK(hipDeviceSynchronize());
    if (rtTransfer) rtTransfer->pause();

    if (rtKernel) rtKernel->start();
    usize numBlocks = n % BlockSize == 0 ? n / BlockSize : n / BlockSize + 1;
    FillKernel<<<numBlocks, BlockSize>>>(
        d_histos, nBins,
        xMin, xMax,
        d_scales, nScales,
        d_resolutions, nResolutions,
        d_defCoords, n
    );
    ERRCHECK(hipDeviceSynchronize());
    ERRCHECK(hipPeekAtLastError());
    if (rtKernel) rtKernel->pause();
}

template class GpuFWM<256, 32768>;
