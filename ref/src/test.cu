#include <cmath>

#include <numeric>
#include <random>
#include <iostream>

// ROOT
#include "TH1.h"

#include "CUDAHelpers.cuh"
#include "RHnCUDA.cuh"

#include "test.h"

#define MAX_ERROR  0.000000000001

using namespace ROOT::Experimental;

template <typename T>
T arraySum(T a[], size_t n)
{
    return std::accumulate(a, a + n, 0);
}

template <typename T>
T arrayProduct(T a[], size_t n)
{
    return std::accumulate(a, a + n, 1, std::multiplies<T>());
}

template <unsigned int Dim>
HistogramTest<Dim>::HistogramTest(double *histogram, double *binEdges, int *binEdgesIdx, int *nBinsAxis,
              double *xMin, double *xMax, double *coords, double *weights,
              bool *mask, size_t bulkSize)
{
    h_histogram = histogram;
    h_binEdges = binEdges;
    h_binEdgesIdx = binEdgesIdx;
    h_nBinsAxis = nBinsAxis;
    h_xMin = xMin;
    h_xMax = xMax;
    h_coords = coords;
    h_weights = weights;
    h_mask = mask;
    this->bulkSize = bulkSize;
}

template <unsigned int Dim>
HistogramTest<Dim>::~HistogramTest()
{
    // TODO: Proper cleanup!
}

template <unsigned int Dim>
void HistogramTest<Dim>::allocateDevice()
{
    ERRCHECK(hipMalloc(&d_histogram, sizeof(double) * arrayProduct(h_nBinsAxis, Dim)));
    if (h_binEdges)
        ERRCHECK(hipMalloc(&d_binEdges, sizeof(double) * arraySum(h_nBinsAxis, Dim)));
    ERRCHECK(hipMalloc(&d_binEdgesIdx, sizeof(int) * Dim));
    ERRCHECK(hipMalloc(&d_nBinsAxis, sizeof(int) * Dim));
    ERRCHECK(hipMalloc(&d_xMin, sizeof(double) * Dim));
    ERRCHECK(hipMalloc(&d_xMax, sizeof(double) * Dim));
    ERRCHECK(hipMalloc(&d_coords, sizeof(double) * bulkSize * Dim));
    ERRCHECK(hipMalloc(&d_weights, sizeof(double) * bulkSize * Dim));
    ERRCHECK(hipMalloc(&d_mask, sizeof(double) * bulkSize * Dim));
}

template <unsigned int Dim>
void HistogramTest<Dim>::transferDevice()
{
    if (h_binEdges)
        ERRCHECK(hipMemcpy(d_binEdges, h_binEdges, sizeof(double) * arraySum(h_nBinsAxis, Dim), hipMemcpyHostToDevice));
    ERRCHECK(hipMemcpy(d_binEdgesIdx, h_binEdgesIdx, sizeof(int) * Dim, hipMemcpyHostToDevice));
    ERRCHECK(hipMemcpy(d_nBinsAxis, h_nBinsAxis, sizeof(int) * Dim, hipMemcpyHostToDevice));
    ERRCHECK(hipMemcpy(d_xMin, h_xMin, sizeof(double) * Dim, hipMemcpyHostToDevice));
    ERRCHECK(hipMemcpy(d_xMax, h_xMax, sizeof(double) * Dim, hipMemcpyHostToDevice));
    ERRCHECK(hipMemcpy(d_coords, h_coords, sizeof(double) * bulkSize * Dim, hipMemcpyHostToDevice));
    ERRCHECK(hipMemcpy(d_weights, h_weights, sizeof(double) * bulkSize * Dim, hipMemcpyHostToDevice));
    ERRCHECK(hipMemcpy(d_mask, h_mask, sizeof(bool) * bulkSize * Dim, hipMemcpyHostToDevice));
}

template <unsigned int Dim>
void HistogramTest<Dim>::run()
{
    HistogramGlobal<double, Dim><<<4, 128>>>(
        d_histogram,
        d_binEdges,
        d_binEdgesIdx,
        d_nBinsAxis,
        d_xMin,
        d_xMax,
        d_coords,
        d_weights,
        d_mask,
        bulkSize
    );

    hipDeviceSynchronize();
    ERRCHECK(hipPeekAtLastError());
}

template <unsigned int Dim>
void HistogramTest<Dim>::transferResult()
{
    ERRCHECK(hipMemcpy(h_histogram, d_histogram, sizeof(double) * arrayProduct(h_nBinsAxis, Dim), hipMemcpyDeviceToHost));
}

template <unsigned int Dim>
void HistogramTest<Dim>::checkResult()
{
    // Comparing against existing ROOT Histogram implementation.
    // This implementation is fixed 1D only, for the time being.

    auto histoROOT = new TH1D(
        "",              // Name
        ";x;y",          // Title
        h_nBinsAxis[0] - 2,
        h_xMin[0],
        h_xMax[0]
    );

    histoROOT->FillN(
        bulkSize,
        h_coords,
        h_weights
    );

    const double *t_histogram = histoROOT->GetArray();

    double maxError = 0.0;
    for (size_t i = 0; i < h_nBinsAxis[0]; ++i) {
        double error = fabsl((h_histogram[i] - t_histogram[i]) / t_histogram[i]);
        if (error > maxError)
            maxError = error;
    }

    if (maxError > MAX_ERROR) {
        std::cerr << "Test failed! Relative maximum error is " << maxError << std::endl;
    }
}

template <unsigned int Dim>
void HistogramTest<Dim>::fullTest()
{
    allocateDevice();
    transferDevice();
    run();
    transferResult();
    checkResult();
}


Single1DFixedUniformHistogramTest::Single1DFixedUniformHistogramTest(
    const int nBins,
    const double xMinVal,
    const double xMaxVal,
    const size_t bulkSize
    ) : HistogramTest(
      new double[nBins],
      nullptr,
      new int[1] {-1},
      new int[1] {nBins},
      new double[1] {xMinVal},
      new double[1] {xMaxVal},
      new double[bulkSize],
      new double[bulkSize],
      new bool[bulkSize],
      bulkSize
    )
{
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> coords_dis(xMinVal, xMaxVal);
    std::uniform_real_distribution<> weight_dis(0.0, 1.0);

    for (size_t i = 0; i < bulkSize; ++i) {
        h_coords[i] = coords_dis(gen);
        h_weights[i] = weight_dis(gen);
        h_mask[i] = true;
    }
}
