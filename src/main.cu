#include <iostream>
#include <numeric>
#include <random>

#include "RHnCUDA.h"

template <typename T>
T arraySum(T a[], size_t n)
{
    return std::accumulate(a, a + n, 0);
}

template <typename T>
T arrayProduct(T a[], size_t n)
{
    return std::accumulate(a, a + n, 1, std::multiplies<T>());
}

using namespace ROOT::Experimental;

template <unsigned int Dim>
class HistogramTest {
protected:
    double *h_histogram;
    double *h_binEdges;  // Can be NULL!
    int    *h_binEdgesIdx;
    int    *h_nBinsAxis;
    double *h_xMin;
    double *h_xMax;
    double *h_coords;
    double *h_weights;
    bool   *h_mask;

    double *d_histogram;
    double *d_binEdges;  // Can be NULL!
    int    *d_binEdgesIdx;
    int    *d_nBinsAxis;
    double *d_xMin;
    double *d_xMax;
    double *d_coords;
    double *d_weights;
    bool   *d_mask;

    size_t bulkSize;

public:
    HistogramTest(double *histogram, double *binEdges, int *binEdgesIdx, int *nBinsAxis,
                  double *xMin, double *xMax, double *coords, double *weights,
                  bool *mask, size_t bulkSize)
    {
        h_histogram = histogram;
        h_binEdges = binEdges;
        h_binEdgesIdx = binEdgesIdx;
        h_nBinsAxis = nBinsAxis;
        h_xMin = xMin;
        h_xMax = xMax;
        h_coords = coords;
        h_weights = weights;
        h_mask = mask;
        this->bulkSize = bulkSize;
    }

    ~HistogramTest()
    {
        // TODO: Proper cleanup!
    }

    void allocateDevice()
    {
        ERRCHECK(hipMalloc(&d_histogram, sizeof(double) * arrayProduct(h_nBinsAxis, Dim)));
        if (h_binEdges)
            ERRCHECK(hipMalloc(&d_binEdges, sizeof(double) * arraySum(h_nBinsAxis, Dim)));
        ERRCHECK(hipMalloc(&d_binEdgesIdx, sizeof(int) * Dim));
        ERRCHECK(hipMalloc(&d_nBinsAxis, sizeof(int) * Dim));
        ERRCHECK(hipMalloc(&d_xMin, sizeof(double) * Dim));
        ERRCHECK(hipMalloc(&d_xMax, sizeof(double) * Dim));
        ERRCHECK(hipMalloc(&d_coords, sizeof(double) * bulkSize * Dim));
        ERRCHECK(hipMalloc(&d_weights, sizeof(double) * bulkSize * Dim));
        ERRCHECK(hipMalloc(&d_mask, sizeof(double) * bulkSize * Dim));
    }

    void transferDevice()
    {
        if (h_binEdges)
            ERRCHECK(hipMemcpy(d_binEdges, h_binEdges, sizeof(double) * arraySum(h_nBinsAxis, Dim), hipMemcpyHostToDevice));
        ERRCHECK(hipMemcpy(d_binEdgesIdx, h_binEdgesIdx, sizeof(int) * Dim, hipMemcpyHostToDevice));
        ERRCHECK(hipMemcpy(d_nBinsAxis, h_nBinsAxis, sizeof(int) * Dim, hipMemcpyHostToDevice));
        ERRCHECK(hipMemcpy(d_xMin, h_xMin, sizeof(double) * Dim, hipMemcpyHostToDevice));
        ERRCHECK(hipMemcpy(d_xMax, h_xMax, sizeof(double) * Dim, hipMemcpyHostToDevice));
        ERRCHECK(hipMemcpy(d_coords, h_coords, sizeof(double) * bulkSize * Dim, hipMemcpyHostToDevice));
        ERRCHECK(hipMemcpy(d_weights, h_weights, sizeof(double) * bulkSize * Dim, hipMemcpyHostToDevice));
        ERRCHECK(hipMemcpy(d_mask, h_mask, sizeof(bool) * bulkSize * Dim, hipMemcpyHostToDevice));
    }

    void run()
    {
        HistogramGlobal<double, Dim><<<4, 128>>>(
            d_histogram,
            d_binEdges,
            d_binEdgesIdx,
            d_nBinsAxis,
            d_xMin,
            d_xMax,
            d_coords,
            d_weights,
            d_mask,
            bulkSize
        );

        hipDeviceSynchronize();
        ERRCHECK(hipPeekAtLastError());
    }

    void transferResult()
    {
        ERRCHECK(hipMemcpy(h_histogram, d_histogram, sizeof(double) * arrayProduct(h_nBinsAxis, Dim), hipMemcpyDeviceToHost));
    }

    void checkResult()
    {
        // TODO: Compare results against a simple CPU implementation.
    }
};

class Single1DFixedUniformHistogramTest : public HistogramTest<1> {
public:
    Single1DFixedUniformHistogramTest(
        const int nBins = 102,
        const double xMinVal = 0.0,
        const double xMaxVal = 100,
        const size_t bulkSize = 1000
        ) : HistogramTest(
          new double[nBins],
          nullptr,
          new int[1] {-1},
          new int[1] {nBins},
          new double[1] {xMinVal},
          new double[1] {xMaxVal},
          new double[bulkSize],
          new double[bulkSize],
          new bool[bulkSize],
          bulkSize
        )
    {
        std::random_device rd;
        std::mt19937 gen(rd());
        std::uniform_real_distribution<> coords_dis(xMinVal, xMaxVal);
        std::uniform_real_distribution<> weight_dis(0.0, 1.0);

        for (size_t i = 0; i < bulkSize; ++i) {
            h_coords[i] = coords_dis(gen);
            h_weights[i] = weight_dis(gen);
            h_mask[i] = true;
        }
    }
};

int main(int argc, char **argv)
{
    auto test = Single1DFixedUniformHistogramTest();
    test.allocateDevice();
    test.transferDevice();
    test.run();
    test.transferResult();

    return 0;
}
